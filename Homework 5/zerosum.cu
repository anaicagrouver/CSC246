// Elapsed Real Time for input-5.txt:
// GPU for this execution time:

// real    0m1.565s
// user    0m0.055s
// sys     0m1.208s

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Input sequence of values.
int *vList;

// Number of values on the list.
int vCount = 0;

// Capacity of the list of values.
int vCap = 0;

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: zerosum [report]\n" );
  exit( 1 );
}

// Read the list of values.
__host__ void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (int *) malloc( vCap * sizeof( int ) );

  // Keep reading as many values as we can.
  int v;
  while ( scanf( "%d", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (int *) realloc( vList, vCap * sizeof( int ) );
    }

    // Store the latest value in the next array slot.
    vList[ vCount++ ] = v;
  }
}

__global__ void checkSum( int vCount, bool report,  int *gpuOut,  int *devList) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //printf("idx %d vCount is %d \n", idx, vCount);
  if ( idx < vCount ) {
    //keeps track of sum
    int sum = 0; 
    //keeps track of count
    int count = 0; 
    for(int i = idx; i < vCount; i++){
      sum += devList[i];
      if(sum == 0){
        count++; 
        if(report)
          printf("%d .. %d\n",idx, i);
      }
    }
    gpuOut[idx] = count; 
}
}

int main( int argc, char *argv[] ) {
  if ( argc < 1 || argc > 2 )
    usage();

  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readList();

  // Allocate space on the device to hold a copy of the sequence.
  int *devList = NULL;
  if ( hipMalloc((void **)&devList, vCount * sizeof(int) ) != hipSuccess )
    fail( "Failed to allocate space for lenght list on device" );
  // Copy the sequence over to the device.
  if ( hipMemcpy( devList, vList, vCount * sizeof(int),
                    hipMemcpyHostToDevice) != hipSuccess )
      fail( "Failed to copy list to device" );
  // Maybe some more code written by you.
  int *gpuOut = NULL;
  hipMalloc( (void **)&gpuOut, vCount * sizeof(int));
  // Block and grid dimensions.
  int threadsPerBlock = 100;
  // Round up for the number of blocks we need.
  int blocksPerGrid = ( vCount + threadsPerBlock - 1 ) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions
  checkSum<<<blocksPerGrid, threadsPerBlock>>>( vCount, report, gpuOut, devList );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // Add code to copy results back to the host and then add up the total
  // number of zero-sum ranges found.
  int result[vCount];
  hipMemcpy(result, gpuOut, vCount * sizeof(int), hipMemcpyDeviceToHost);

  //calculating total 
  int total = 0;
  for(int i = 0; i < vCount; i++)
    total += result[i];
  printf( "Total: %d\n", total );

  // Free memory on the device and the host.
  hipFree( devList );
  free( vList );
  hipFree( gpuOut );
  hipDeviceReset();
  return 0;
}
